#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed: Shubh Patel Date: 04/5/2024                                    */
/*                                                                        */
/*                                                                        */
/* CPSC 677 CUDA Histogram lab, Version 1.02, Spring 2024.                */

#include "helper_timer.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM_BINS 4096

__global__ void histogram_kernel(unsigned int* input, unsigned int* bins,
    unsigned int num_elements,
    unsigned int num_bins)
{
    //@@ Declare and clear privatized bins
    __shared__ unsigned int histo_private[32];

    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (globalIndex < num_bins)
    {
        histo_private[threadIdx.x] = 0;
    }
    __syncthreads();

    //@@ Compute histogram
    if (globalIndex < num_elements)
    {
        atomicAdd(&histo_private[input[globalIndex]], 1);
    }
    __syncthreads();

    //@@ Commit to global memory
    if (globalIndex < num_bins)
    {
        atomicAdd(&bins[globalIndex], histo_private[threadIdx.x]);
    }
}

__global__ void convert_kernel(unsigned int* bins, unsigned int num_bins)
{
    //@@ Ensure bin values are not too large
    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (globalIndex < num_bins)
    {
        if (bins[globalIndex] > 127)
        {
            bins[globalIndex] = 127;
        }
    }
}

void histogram(unsigned int* input, unsigned int* bins,
    unsigned int num_elements, unsigned int num_bins)
{

    //@@ zero out bins
    hipMemset(bins, 0, sizeof(unsigned int) * num_bins);

    // Initilize the grid and block dimensions
    dim3 gridDim(NUM_BINS/32, 1, 1);
    dim3 blockDim(32, 1, 1);

    //@@ Launch histogram_kernel on the bins
    {
        histogram_kernel<<<gridDim, blockDim>>>(input, bins, num_elements, num_bins);
        hipDeviceSynchronize();
    }

    //@@ Launch convert_kernel on the bins
    {
        convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
        hipDeviceSynchronize();
    }
}

int main(int argc, char* argv[])
{
    int inputLength, outputLength;
    unsigned int* hostInput;
    unsigned int* hostBins;
    unsigned int* expectedOutput;
    unsigned int* deviceInput;
    unsigned int* deviceBins;

    FILE *infile, *outfile;
    StopWatchLinux stw;
    unsigned int blog = 1;

    // Import host input data
    stw.start();
    if ((infile = fopen("input.raw", "r")) == NULL)
    {
        printf("Cannot open input.raw.\n");
        exit(EXIT_FAILURE);
    }
    fscanf(infile, "%i", &inputLength);
    hostBins = (unsigned int*)malloc(NUM_BINS * sizeof(unsigned int));
    hostInput = (unsigned int*)malloc(sizeof(unsigned int) * inputLength);
    for (int i = 0; i < inputLength; i++)
        fscanf(infile, "%i", &hostInput[i]);
    fclose(infile);
    stw.stop();
    printf("Importing data and creating memory on host: %f ms\n", stw.getTime());

    if (blog)
        printf("*** The input length is %i\n", inputLength);
    if (blog)
        printf("*** The number of bins is %i\n", NUM_BINS);

    stw.reset();
    stw.start();

    //@@ Allocate GPU memory here
    hipMalloc((void**) &deviceInput, sizeof(unsigned int) * inputLength);
    hipMalloc((void**) &deviceBins, sizeof(unsigned int) * NUM_BINS);

    hipDeviceSynchronize();

    stw.stop();
    printf("Allocating GPU memory: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, sizeof(unsigned int) * inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceBins, hostBins, sizeof(unsigned int) * NUM_BINS, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    stw.stop();
    printf("Copying input memory to the GPU: %f ms\n", stw.getTime());

    // Launch kernel
    // ----------------------------------------------------------
    if (blog)
        printf("*** Launching kernel");

    stw.reset();
    stw.start();

    histogram(deviceInput, deviceBins, inputLength, NUM_BINS);

    stw.stop();
    printf("Performing CUDA computation: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostBins, deviceBins, sizeof(unsigned int) * NUM_BINS, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    stw.stop();
    printf("Copying output memory to the CPU: %f ms\n", stw.getTime());

    stw.reset();
    stw.start();

    //@@ Free the GPU memory here
    hipFree(deviceInput);
    hipFree(deviceBins);

    stw.stop();
    printf("Freeing GPU Memory: %f ms\n", stw.getTime());

    // Verify correctness
    // -----------------------------------------------------

    if ((outfile = fopen("output.raw", "r")) == NULL)
    {
        printf("Cannot open output.raw.\n");
        exit(EXIT_FAILURE);
    }
    fscanf(outfile, "%i", &outputLength);
    expectedOutput = (unsigned int*)malloc(sizeof(unsigned int) * outputLength);
    for (int i = 0; i < outputLength; i++)
        fscanf(outfile, "%i", &expectedOutput[i]);
    fclose(outfile);

    int test = 1;
    for (int i = 0; i < outputLength; i++)
        test = test && (expectedOutput[i] == hostBins[i]);

    if (test)
        printf("Results correct.\n");
    else
        printf("Results incorrect.\n");

    free(hostBins);
    free(hostInput);
    free(expectedOutput);
    return 0;
}
